#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###

#include "aux.h"
#include <iostream>
#include <stdio.h>
using namespace std;

// uncomment to use the camera
// #define CAMERA

void Print1D(float* x, int w, int h, int l) {
    for (int k = 0; k < l; k++) {
        cout << "_____ Level: " << k << " _____" << endl;
        for (int i = 0; i < h; i++) {
            for (int j = 0; j < w; j++) {
                cout << "(";
                // for (int c = 0; c < Dimension(); c++) {
                    // printf("%g ", Get(i, j, k, c));
                    cout << x[j + i * w + k * h * w] << "";
                    // cout << (int)(x[j + i * w + k * h * w]*255) << "";
                // }
                cout << ")" << "";
            }
            cout << endl;
        }
        cout << "_____          _____" << endl << endl;
    }
}

void Print3D(float* x1, float* x2, float* x3, int w, int h, int l) {
    for (int k = 0; k < l; k++) {
        cout << "_____ Level: " << k << " _____" << endl;
        for (int i = 0; i < h; i++) {
            for (int j = 0; j < w; j++) {
                // printf("(%2.g,%2.g,%2.g)", x1[j + i * w + k * h * w], x2[j + i * w + k * h * w], x3[j + i * w + k * h * w]);
                cout << "(";
                cout << x1[j + i * w + k * h * w] << " " << x2[j + i * w + k * h * w] << " " << x3[j + i * w + k * h * w] << "";
                cout << ")" << "";
            }
            // printf("\n");
            cout << endl;
        }
        cout << "_____          _____" << endl << endl;
    }
}

void Print4D(float* x1, float* x2, float* x3, int w, int h, int l, int p) {
    for (int k = 0; k < 1; k++) {
        cout << "_____ Level: " << k << " _____" << endl;
        for (int i = 0; i < h; i++) {
            for (int j = 0; j < w; j++) {
                cout << "(";
                cout << x1[j + i * w + k * h * w + p * w * h * l] << " " << x2[j + i * w + k * h * w + p * w * h * l] << " " << x3[j + i * w + k * h * w + p * w * h * l] << "";
                cout << ")" << "";
                // printf("(%2.g,%2.g,%2.g)", x1[j + i * w + k * h * w], x2[j + i * w + k * h * w], x3[j + i * w + k * h * w]);
            }
            // printf("\n");
            cout << endl;
        }
        cout << "_____          _____" << endl << endl;
    }
}

__device__ float l2Norm(float x1, float x2)
{
    return sqrtf(x1*x1 + x2*x2);
}

__device__ float bound(float x1, float x2, float lambda, float k, float L, float f)
{
    return 0.25f * (x1*x1 + x2*x2) - lambda * pow(k / L - f, 2);
}

__device__ float interpolate(float k, float uk0, float uk1, float l)
{
    return (k + (0.5 - uk0) / (uk1 - uk0)) / (l-1);
}

__device__ void on_parabola(float* u1, float* u2, float* u3, float x1, float x2, float x3, float f, float L, float lambda, float k, int j)
{
    float y = x3 + lambda * pow(k / L - f, 2);
    float norm = l2Norm(x1, x2);
    float v = 0.f;
    float a = 2.f * 0.25f * norm;
    float b = 2.f / 3.f * (1.f - 2.f * 0.25f * y);
    float d = b < 0 ? (a - pow(sqrt(-b), 3)) * (a + pow(sqrt(-b), 3)) : a*a + b*b*b;
    float c = pow((a + sqrt(d)), 1.f/3.f);
    if (d >= 0) {
        v = c == 0 ? 0.f : c - b / c;
    } else {
        v = 2.f * sqrt(-b) * cos((1.f / 3.f) * acos(a / (pow(sqrt(-b), 3))));
    }
    u1[j] = norm == 0 ? 0.f : (v / (2.0 * 0.25f)) * x1 / norm;
    u2[j] = norm == 0 ? 0.f : (v / (2.0 * 0.25f)) * x2 / norm;
    u3[j] = bound(u1[j], u2[j], lambda, k, L, f);
}

// /**
// * @brief Computes the orthogonal projection of (x0, y0) onto the epigraph of
// *        the parabola y >= \alpha x^2.
// */
// template<typename T>
// inline __device__ void ProjectParabolaSimple(const T& x0,
//                                             const T& y0,
//                                             const T& alpha,
//                                             T& x,
//                                             T& y)
// {
//  // nothing to do?
//  if(y0 >= alpha * (x0 * x0)) {
//    x = x0;
//    y = y0;
//  }
//  else {
//    const T a = 2. * alpha * fabs(x0);
//    const T b = 2. * (1. - 2. * alpha * y0) / 3.;
//    T d, v;
   
//    if(b < 0) {
//      const T sq = powf(-b, 3. / 2.);
//      d = (a - sq) * (a + sq);      
//    }
//    else {
//      d = a * a + b * b * b;
//    }
   
//    if(d >= 0) {
//      const T c = powf(a + sqrtf(d), 1. / 3.);
//      v = c - b / c;
//    }
//    else {
//      v = 2 * sqrtf(-b) * cos(acos(a / powf(-b, 3. / 2.)) / 3.);
//    }

//    if(x0 > 0)
//      x = v / (2. * alpha);
//    else if(x0 < 0)
//      x = -v / (2. * alpha);
//    else
//      x = 0;

//    y = alpha * x * x;
//  }
// }

// /**
// * @brief Computes orthogonal projection of (x0, y0) onto the epigraph of the
// *        parabola y >= p * x^2 + q * x + r.
// */
// template<typename T>
// inline __device__ void ProjectParabolaGeneral(const T& x0,
//                                              const T& y0,
//                                              const T& p,
//                                              const T& q,
//                                              const T& r,
//                                              T& x,
//                                              T& y)
// {
//  T tildex;
//  T tildey;
 
//  ProjectParabolaSimple<T>(
//      x0 + q / (2. * p),
//      y0 + q * q / (4. * p) - r,
//      p,
//      tildex,
//      tildey);
 
//  x = tildex - q / (2. * p);
//  y = tildey - q * q / (4. * p) + r;
// }
__global__ void project_on_parabola(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float* img, float L, float lambda, int k, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int index = x + w * y;
        int i = x + w * y + w * h * z + (k-1) * w * h * l;
        int j = x + w * y + w * h * z + k * w * h * l;

        float f = img[index];
        float x1 = u1[i] - v1[j];
        float x2 = u2[i] - v2[j];
        float x3 = u3[i] - v3[j];
        float bound_val = bound(x1, x2, lambda, z+1.f, L, f);

        if (x3 < bound_val) {
            on_parabola(u1, u2, u3, x1, x2, x3, f, L, lambda, z+1.f, j);
            // ProjectParabolaGeneral(0.288675f, 0.f, 0.25f, 0.f, bound_val, x1, x3);
        } else {
            u1[j] = x1;
            u2[j] = x2;
            u3[j] = x3;
        }
    }
}

__device__ float partial(float x)
{
    return 0.5 * x;
}

__global__ void soft_shrinkage(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float nu, int k1, int k2, int P, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    const float K = (float)(k2 - k1 + 1);

    if (x < w && y < h)
    {
        int i, j;
        float s1 = 0.f;
        float s2 = 0.f;
        float s01 = 0.f;
        float s02 = 0.f;
        float x1 = 0.f;
        float x2 = 0.f;

        for (int k = k1; k <= k2; k++)
        {
            i = x + w * y + k * w * h + (P-1) * w * h * l;
            j = x + w * y + k * w * h + P * w * h * l;
            x1 = u1[i] - v1[j];
            x2 = u2[i] - v2[j];
            s01 += x1;
            s02 += x2;
        }

        float norm = l2Norm(s01, s02);

        s1 = norm <= nu ? s01 : (nu * s01 / norm);
        s2 = norm <= nu ? s02 : (nu * s02 / norm);

        for (int k = 0; k < l; k++)
        {
            i = x + w * y + k * w * h + (P-1) * w * h * l;
            j = x + w * y + k * w * h + P * w * h * l;
            x1 = u1[i] - v1[j];
            x2 = u2[i] - v2[j];
            if (k >= k1 && k <= k2) {
                u1[j] = x1 + (s1 - s01) / K;
                u2[j] = x2 + (s2 - s02) / K;
            } else {
                u1[j] = x1;
                u2[j] = x2;
            }
            u3[j] = u3[i] - v3[j];
        }
    }
}

__global__ void init(float* xbar, float* xcur, float* xn, float* y1, float* y2, float* y3, float* img, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h)
    {
        float img_val = img[x + w * y];
        for (int k = 0; k < l; k++)
        {
            int index = x + w * y + k * w * h;
            xn[index] = img_val;
            xcur[index] = img_val;
            xbar[index] = img_val;
            y1[index] = 0.f;
            y2[index] = 0.f;
            y3[index] = 0.f;
        }
    }
}

__global__ void isosurface(float* img, float* xbar, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h)
    {
        float val = 0.f;
        float uk0 = 0.f;
        float uk1 = 0.f;

        for (int k = 0; k < l-1; k++)
        {
            uk0 = xbar[x + w * y + k * w * h];
            uk1 = xbar[x + w * y + (k+1) * w * h];
            if (uk0 > 0.5 && uk1 <= 0.5) {
                val = interpolate(k, uk0, uk1, l);
                break;
            } else {
                val = 1.f;
            }
        }
        
        img[x + w * y] = val;
    }
}

// __global__ void isosurface(float* img, float* xbar, int w, int h, int l)
// {
//     int x = threadIdx.x + blockDim.x * blockIdx.x;
//     int y = threadIdx.y + blockDim.y * blockIdx.y;
    
//     int i = x + w * y;
//     int k = 0;
//     if (x < w && y < h)
//     {
//         float val = 0.f;
//         float uk0 = 0.f;
//         float uk1 = 0.f;
        
//         while (k < l-1)
//         {
//             uk0 = xbar[i + k * w * h];
//             uk1 = xbar[i + (k+1) * w * h];
//             if (uk0 > 0.5 && uk1 <= 0.5)
//             {
//                 val = interpolate(k, uk0, uk1, l);
//                 k = l;
//                 break;
//             } else {
//                 k++;
//             }
//         }

//         if (k == l)
//             img[i] = val;
//         else
//             img[i] = 1.f;
//     }
// }

__global__ void set_y(float* y1, float* y2, float* y3, float* u1, float* u2, float* u3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j = x + w * y + w * h * z + (p-1) * w * h * l;
        y1[i] = u1[j];
        y2[i] = u2[j];
        y3[i] = u3[j];
    }
}

__global__ void set_u_v(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float* dx, float* dy, float* dz, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j;
        for (int k = 0; k < p; k++)
        {
            j = x + w * y + w * h * z + k * w * h * l;

            u1[j] = k < p-1 ? 0.f : dx[i];
            u2[j] = k < p-1 ? 0.f : dy[i];
            u3[j] = k < p-1 ? 0.f : dz[i];

            // u1[j] = dx[i];
            // u2[j] = dy[i];
            // u3[j] = dz[i];

            v1[j] = 0.f;
            v2[j] = 0.f;
            v3[j] = 0.f;
            
        }
    }
}

__global__ void update_v(float* v1, float* v2, float* v3, float* u1, float* u2, float* u3, int w, int h, int l, int k)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z + k * w * h * l;
        int j = x + w * y + w * h * z + (k-1) * w * h * l;
        v1[i] = u1[i] - (u1[j] - v1[i]);
        v2[i] = u2[i] - (u2[j] - v2[i]);
        v3[i] = u3[i] - (u3[j] - v3[i]);
    }
}

__global__ void set_u_zero(float* u1, float* u2, float* u3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j = x + w * y + w * h * z + (p-1) * w * h * l;
        u1[i] = u1[j];
        u2[i] = u2[j];
        u3[i] = u3[j];
    }
}

__global__ void set_tmp_u(float* tmp1, float* tmp2, float* tmp3, float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    if (x < w && y < h && z < l)
    {
        tmp1[x + w * y + w * h * z] = u1[x + w * y + w * h * z] - v1[x + w * y + w * h * z + p * w * h * l];
        tmp2[x + w * y + w * h * z] = u2[x + w * y + w * h * z] - v2[x + w * y + w * h * z + p * w * h * l];
        tmp3[x + w * y + w * h * z] = u3[x + w * y + w * h * z] - v3[x + w * y + w * h * z + p * w * h * l];
    }
}

__global__ void calc_norm(float* r1, float* r2, float* r3, float* u1, float* u2, float* u3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        r1[i] = u1[i] - u1[i + (p-1) * w * h * l];
        r2[i] = u2[i] - u2[i + (p-1) * w * h * l];
        r3[i] = u3[i] - u3[i + (p-1) * w * h * l];    
    }
}

__global__ void set_r(float* r1, float* r2, float* r3, float* u1, float* u2, float* u3, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    if (x < w && y < h && z < l)
    {
        r1[x + w * y + w * h * z] = u1[x + w * y + w * h * z];
        r2[x + w * y + w * h * z] = u2[x + w * y + w * h * z];
        r3[x + w * y + w * h * z] = u3[x + w * y + w * h * z];
    }
}

__global__ void gradient(float* dx, float* dy, float* dz, float* y1, float* y2, float* y3, float* xbar, float sigma, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        float val = xbar[x + w * y + w * h * z];
        float x1 = (x+1<w) ? (xbar[(x+1) + w * y + w * h * z] - val) : 0.f;
        float x2 = (y+1<h) ? (xbar[x + w * (y+1) + w * h * z] - val) : 0.f;
        float x3 = (z+1<l) ? (xbar[x + w * y + w * h * (z+1)] - val) : 0.f;
        dx[x + w * y + w * h * z] = y1[x + w * y + w * h * z] + sigma * x1;
        dy[x + w * y + w * h * z] = y2[x + w * y + w * h * z] + sigma * x2;
        dz[x + w * y + w * h * z] = y3[x + w * y + w * h * z] + sigma * x3;
    }
}

// __global__ void gradient(float* dx, float* dy, float* dz, float* y1, float* y2, float* y3, float* xbar, float sigma, int w, int h, int l)
// {
//     int x = threadIdx.x + blockDim.x * blockIdx.x;
//     int y = threadIdx.y + blockDim.y * blockIdx.y;
//     int z = threadIdx.z + blockDim.z * blockIdx.z;
    
//     int i = x + w * y + w * h * z;
//     int xi = (x+1) + w * y + w * h * z;
//     int yi = x + w * (y+1) + w * h * z;
//     int zi = x + w * y + w * h * (z+1);

//     if (x < w && y < h && z < l)
//     {
//         float val = xbar[i];
//         dx[i] = y1[i] + sigma * (xbar[min(max(0, xi), w-1)] - val);
//         dy[i] = y2[i] + sigma * (xbar[min(max(0, yi), h-1)] - val);
//         dz[i] = y3[i] + sigma * (xbar[min(max(0, zi), l-1)] - val);
//     }
// }

__global__ void clipping(float* xn, float* xcur, float* y1, float* y2, float* y3, float tau, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        float d1 = y1[x + w * y + w * h * z] - (x>0 ? y1[(x-1) + w * y + w * h * z] : 0.f);
        float d2 = y2[x + w * y + w * h * z] - (y>0 ? y2[x + w * (y-1) + w * h * z] : 0.f);
        float d3 = y3[x + w * y + w * h * z] - (z>0 ? y3[x + w * y + w * h * (z-1)] : 0.f);
        float val = xcur[x + w * y + w * h * z] + tau * (d1 + d2 + d3);
        if (z == 0) {
            xn[x + w * y + w * h * z] = 1.f;
        } else if (z == l-1) {
            xn[x + w * y + w * h * z] = 0.f;
        } else {
            xn[x + w * y + w * h * z] = fmin(1.f, fmax(0.f, val));
        }
    }
}

// __global__ void clipping(float* xn, float* y1, float* y2, float* y3, float tau, int w, int h, int l)
// {
//     int x = threadIdx.x + blockDim.x * blockIdx.x;
//     int y = threadIdx.y + blockDim.y * blockIdx.y;
//     int z = threadIdx.z + blockDim.z * blockIdx.z;
    
//     int i = x + w * y + w * h * z;
//     int xi = (x-1) + w * y + w * h * z;
//     int yi = x + w * (y-1) + w * h * z;
//     int zi = x + w * y + w * h * (z-1);
    
//     float d1, d2, d3, val;

//     if (x < w && y < h && z < l)
//     {
//         d1 = y1[i] - y1[min(max(0, xi), w-1)];
//         d2 = y2[i] - y2[min(max(0, yi), h-1)];
//         d3 = y3[i] - y3[min(max(0, zi), l-1)];
//         val = xn[i] + tau * (d1 + d2 + d3);
        
//         if (z == 0) {
//             xn[i] = 1.f;
//         } else if (z == l-1) {
//             xn[i] = 0.f;
//         } else {
//             xn[i] = fmin(1.f, fmax(0.f, val));
//         }
//     }
// }

__global__ void extrapolate(float* xbar, float* xcur, float* xn, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    int i = x + w * y + w * h * z;

    if (x < w && y < h && z < l) {
        xbar[i] = 2 * xn[i] - xcur[i];
        xcur[i] = xn[i];
        // float val = xn[i];
        // xbar[i] = 2.f * val - xcur[i];
    }
}

int main(int argc, char **argv)
{
    // double TAU = 1.0 / sqrt(12.0);
    // double a = TAU / 2.0;
    // double b = 2.0 / 3.0 - 2.0 / (60.0 * 12.0);
    // double d = a*a + b*b*b;
    // double c = pow((a + sqrt(d)), 1.f/3.f);
    // double v = c - b/c;
    // cout << "x1 = " << 2.0 * v << " x2 = " << 0 << " x3 = " << (2.0*v)*(2.0*v)/4.0 - 0.1*(1.0/sqrt(12.0))*(1.0/sqrt(12.0)) << endl;
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }

#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;

    // number of computation repetitions to get a better run time measurement
    int dykstra = 1;
    getParam("dykstra", dykstra, argc, argv);
    cout << "dykstra: " << dykstra << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = true;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // load the input image as grayscale if "-gray" is specifed
    int level = 16;
    getParam("level", level, argc, argv);
    cout << "level: " << level << endl;

    // load the input image as grayscale if "-gray" is specifed
    float L = sqrtf(12);
    getParam("L", L, argc, argv);
    cout << "L: " << L << endl;

    // load the input image as grayscale if "-gray" is specifed
    // float tau = 1000;
    float tau = 1.f/L;
    getParam("tau", tau, argc, argv);
    cout << "tau: " << tau << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    float sigma = 1.f/(L*L*tau);
    getParam("sigma", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;

    // load the input image as grayscale if "-gray" is specifed
    float lambda = 0.1;
    getParam("lambda", lambda, argc, argv);
    cout << "lambda: " << lambda << endl;

    // load the input image as grayscale if "-gray" is specifed
    float nu = 5.f;
    getParam("nu", nu, argc, argv);
    cout << "nu: " << nu << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = 1;  // number of channels
    // int nc = mIn.channels();  // number of channels
    int dim = w*h*nc;
    int size = w*h*nc*level;
    int projections = level * (level+1) / 2 + 1 + 1;
    int nbytes = size*sizeof(float);
    int nbyted = dim*sizeof(float);
    int nbytep = projections*size*sizeof(float);
    cout << "image: " << w << " x " << h << endl;

    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    // allocate raw input image array
    float* h_imgIn  = new float[(size_t)dim];
    float* h_imgOut = new float[(size_t)dim];
    float* h_x1 = new float[(size_t)size];
    float* h_x2 = new float[(size_t)size];
    float* h_x3 = new float[(size_t)size];
    float* h_r = new float[(size_t)size];
    float* h_u1 = new float[(size_t)projections*size];
    float* h_u2 = new float[(size_t)projections*size];
    float* h_u3 = new float[(size_t)projections*size];

    // allocate raw input image for GPU
    float* d_imgIn; hipMalloc(&d_imgIn, nbyted); CUDA_CHECK;
    float* d_imgOut;hipMalloc(&d_imgOut, nbyted); CUDA_CHECK;

    float* d_x; hipMalloc(&d_x, nbytes); CUDA_CHECK;
    float* d_xbar; hipMalloc(&d_xbar, nbytes); CUDA_CHECK;
    float* d_xcur; hipMalloc(&d_xcur, nbytes); CUDA_CHECK;

    float* d_delX; hipMalloc(&d_delX, nbytes); CUDA_CHECK;
    float* d_delY; hipMalloc(&d_delY, nbytes); CUDA_CHECK;
    float* d_delZ; hipMalloc(&d_delZ, nbytes); CUDA_CHECK;

    float* d_y1; hipMalloc(&d_y1, nbytes); CUDA_CHECK;
    float* d_y2; hipMalloc(&d_y2, nbytes); CUDA_CHECK;
    float* d_y3; hipMalloc(&d_y3, nbytes); CUDA_CHECK;

    float* d_u1; hipMalloc(&d_u1, nbytep); CUDA_CHECK;
    float* d_u2; hipMalloc(&d_u2, nbytep); CUDA_CHECK;
    float* d_u3; hipMalloc(&d_u3, nbytep); CUDA_CHECK;

    float* d_v1; hipMalloc(&d_v1, nbytep); CUDA_CHECK;
    float* d_v2; hipMalloc(&d_v2, nbytep); CUDA_CHECK;
    float* d_v3; hipMalloc(&d_v3, nbytep); CUDA_CHECK;

    // float* d_r1; hipMalloc(&d_r1, nbytep); CUDA_CHECK;
    // float* d_r2; hipMalloc(&d_r2, nbytep); CUDA_CHECK;
    // float* d_r3; hipMalloc(&d_r3, nbytep); CUDA_CHECK;

    // size_t available, total;
    // hipMemGetInfo(&available, &total);
    // cout << available << " " << total << endl;

    // alloc GPU memory

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;

#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (h_imgIn, mIn);

    // copy host memory
    hipMemcpy(d_imgIn, h_imgIn, nbyted, hipMemcpyHostToDevice); CUDA_CHECK;

    // launch kernel
    // dim3 block = dim3(32, 8, nc);
    // dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (nc + block.z - 1) / block.z);
    dim3 block = dim3(32, 8, 4);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (level + block.z - 1) / block.z);
    dim3 block_iso = dim3(32, 8, 1);
    dim3 grid_iso = dim3((w + block_iso.x - 1) / block_iso.x, (h + block_iso.y - 1) / block_iso.y, 1);

    Timer timer; timer.start();

    int count_p = projections;
    float sum = 0.f;

    init <<<grid_iso, block_iso>>> (d_xbar, d_xcur, d_x, d_y1, d_y2, d_y3, d_imgIn, w, h, level);
    // hipMemcpy(h_x1, d_xcur, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
    // Print1D(h_x1, w, h, level);

    for (int i = 1; i <= repeats; i++)
    {
        // cout << "Step " << i << " of " << repeats << " with count_p = " << count_p << endl;
        
        gradient <<<grid, block>>> (d_delX, d_delY, d_delZ, d_y1, d_y2, d_y3, d_xbar, sigma, w, h, level);
        // hipMemcpy(h_x1, d_delY, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        // hipMemcpy(h_x2, d_delX, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        // hipMemcpy(h_x3, d_delZ, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        // Print3D(h_x1, h_x2, h_x3, w, h, level);
        // Print1D(h_x1, w, h, level);

        set_u_v <<<grid, block>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, d_delX, d_delY, d_delZ, w, h, level, projections);
        // hipMemcpy(h_u1, d_u2, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
        // hipMemcpy(h_u2, d_u1, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
        // hipMemcpy(h_u3, d_u3, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
        // Print4D(h_u1, h_u2, h_u3, w, h, level, 0);
        // Print3D(h_u1, h_u2, h_u3, w, h, level);
        
        for (int j = 0; j < dykstra; j++)
        {            
            count_p = 1;
            set_u_zero <<<grid, block>>> (d_u1, d_u2, d_u3, w, h, level, projections);
            // Print3D(h_u1, h_u2, h_u3, w, h, level);
            
            // hipMemcpy(h_u1, d_u2, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // hipMemcpy(h_u2, d_u1, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // hipMemcpy(h_u3, d_u3, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // Print4D(h_u1, h_u2, h_u3, w, h, level, 0);
            // parabola y >= p * x^2 + q * x + r.
            project_on_parabola <<<grid, block>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, d_imgIn, L, lambda, count_p, w, h, level);
            update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, count_p);
            // hipMemcpy(h_u1, d_u2, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // hipMemcpy(h_u2, d_u1, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // hipMemcpy(h_u3, d_u3, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // Print4D(h_u1, h_u2, h_u3, w, h, level, count_p);
            count_p++;
            // Print4D(h_u1, h_u2, h_u3, w, h, level, 1);
            // Print4D(h_u1, h_u2, h_u3, w, h, level, 2);
            
            for (int k1 = 0; k1 < level; k1++)
            {
                for (int k2 = k1; k2 < level; k2++)
                {
                    soft_shrinkage <<<grid_iso, block_iso>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, nu, k1, k2, count_p, w, h, level);
                    update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, count_p);
                    // hipMemcpy(h_u1, d_u2, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
                    // hipMemcpy(h_u2, d_u1, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
                    // hipMemcpy(h_u3, d_u3, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
                    // Print4D(h_u1, h_u2, h_u3, w, h, level, count_p);
                    count_p++;
                }
            }

            // for (int k = 1; k < projections; k++)
            // {
            //     update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, k);
            // }

            // calc_norm <<<grid, block>>> (d_r1, d_r2, d_r3, d_u1, d_u2, d_u3, w, h, level, projections);
            // hipMemcpy(h_u1, d_r2, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // hipMemcpy(h_u2, d_r1, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // hipMemcpy(h_u3, d_r3, nbytep, hipMemcpyDeviceToHost); CUDA_CHECK;
            // Print4D(h_u1, h_u2, h_u3, w, h, level, 0);
            // sum = 0.f;
            // for (int i = 0; i < size; i++)
            // {
            //     sum += pow((h_u1[i] + h_u2[i] + h_u3[i]), 2);
            // }
        }
        // cout << "NORM = " << sqrtf(sum) << endl;
        
        set_y <<<grid, block>>> (d_y1, d_y2, d_y3, d_u1, d_u2, d_u3, w, h, level, projections);
        
        hipMemcpy(h_x1, d_y1, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(h_x2, d_y2, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(h_x3, d_y3, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;

        sum = 0.f;
        for (int kx = 0; kx < level; kx++)
        {
            for (int ix = 0; ix < h; ix++)
            {
                for (int jx = 0; jx < w; jx++)
                {
                    float x1 = h_x1[jx + w * ix + w * h * kx] - (jx>0 ? h_x1[(jx-1) + w * ix + w * h * kx] : 0.f);
                    float x2 = h_x2[jx + w * ix + w * h * kx] - (ix>0 ? h_x2[jx + w * (ix-1) + w * h * kx] : 0.f);
                    float x3 = h_x3[jx + w * ix + w * h * kx] - (kx>0 ? h_x3[jx + w * ix + w * h * (kx-1)] : 0.f);
                    float d = x1+x2+x3;
                    if (d <= 0) {
                        sum += 1.f;
                        h_r[jx + w * ix + w * h * kx] = 0.f;
                    } else {
                        h_r[jx + w * ix + w * h * kx] = 1.f;
                    }
                }
            }
        }
        printf("%d %f\n", i, sqrtf(sum));

        // Print3D(h_x1, h_x2, h_x3, w, h, level);
        clipping <<<grid, block>>> (d_x, d_xcur, d_y1, d_y2, d_y3, tau, w, h, level);
        // clipping <<<grid, block>>> (d_x, d_xcur, d_delX, d_delY, d_delZ, tau, w, h, level);
        // hipMemcpy(h_x1, d_xbar, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        // Print1D(h_x1, w, h, level);

        extrapolate <<<grid, block>>> (d_xbar, d_xcur, d_x, w, h, level);
        // if (i%20 == 0) {
        //     isosurface <<<grid_iso, block_iso>>> (d_imgOut, d_x, w, h, level);
        //     hipMemcpy(h_imgOut, d_imgOut, nbyted, hipMemcpyDeviceToHost); CUDA_CHECK;
        //     convert_layered_to_mat(mOut, h_imgOut);
        //     showImage("Output", mOut, 100+w+40, 100);
        //     cv::waitKey(0);
        // }
        // Print1D(h_imgOut, w, h, 1);
    }
    // hipMemcpy(h_x1, d_x, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
    // Print1D(h_x1, w, h, level);

    isosurface <<<grid_iso, block_iso>>> (d_imgOut, d_x, w, h, level);
    // hipMemcpy(h_imgOut, d_imgOut, nbyted, hipMemcpyDeviceToHost); CUDA_CHECK;
    // Print1D(h_imgOut, w, h, 1);

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;

    hipMemcpy(h_imgOut, d_imgOut, nbyted, hipMemcpyDeviceToHost); CUDA_CHECK;

    // free GPU memory
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    
    hipFree(d_x); CUDA_CHECK;
    hipFree(d_xbar); CUDA_CHECK;
    hipFree(d_xcur); CUDA_CHECK;

    hipFree(d_delX); CUDA_CHECK;
    hipFree(d_delY); CUDA_CHECK;
    hipFree(d_delZ); CUDA_CHECK;

    hipFree(d_y1); CUDA_CHECK;
    hipFree(d_y2); CUDA_CHECK;
    hipFree(d_y3); CUDA_CHECK;

    hipFree(d_u1); CUDA_CHECK;
    hipFree(d_u2); CUDA_CHECK;
    hipFree(d_u3); CUDA_CHECK;

    hipFree(d_v1); CUDA_CHECK;
    hipFree(d_v2); CUDA_CHECK;
    hipFree(d_v3); CUDA_CHECK;

    // hipFree(d_r1); CUDA_CHECK;
    // hipFree(d_r2); CUDA_CHECK;
    // hipFree(d_r3); CUDA_CHECK;

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, h_imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] h_imgIn;
    delete[] h_imgOut;
    delete[] h_x1;
    delete[] h_x2;
    delete[] h_x3;
    delete[] h_u1;
    delete[] h_u2;
    delete[] h_u3;
    delete[] h_r;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}